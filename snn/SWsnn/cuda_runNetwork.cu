#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <string.h>
#include <assert.h>
#include "swstruct.h"
#include "mygpu.h"

//#include"./common/book.h"
//#include <unistd.h>        // for sleep()
//#define NBLOCK 64
//#define NTHREAD 128 

/*swInfo_t *cu_swInfo;
neurInfo_t *cu_nInfo;
synInfo_t *cu_sInfo;*/

/**************for delay*************************/
/*spikeTime_t *cu_firingTable;
float *cu_ringBuffer;*/
/*************************************************/

// 检查显卡错误
void checkError() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << std::endl;
    }
}
void initSW(snnInfo_t *sInfo, swInfo_t *swInfo){//????
  hipHostMalloc(&sInfo->nInfoHost, sInfo->gSize*sizeof(neurInfo_t));  // 内存空间
  hipMalloc(&sInfo->nInfoDevice, sInfo->gSize*sizeof(neurInfo_t));  // 显存空间
  hipMemcpy(sInfo->nInfoDevice, sInfo->nInfoHost,
    sInfo->gSize*sizeof(neurInfo_t), hipMemcpyHostToDevice);  // 拷贝数据
  
	//以下两个变量(神经元、突触)可有可无
	//hipMalloc 神经元数据 
	//显存变量：sInfo->nInfoDevice 对应主存变量：sInfo->nInfoHost (类型neurInfo_t*)
	//
	//空间大小sInfo->gSize*sizeof(neurInfo_t)
	
	//hipMemcpy 以上主存到显存数据拷贝
	
  hipHostMalloc(&sInfo->sInfoHost,
    sInfo->preN*sInfo->Ndelay*sInfo->MaxN*NTh*sizeof(synInfo_t));  // 内存空间
  hipMalloc(&sInfo->sInfoDevice,
    sInfo->preN*sInfo->Ndelay*sInfo->MaxN*NTh*sizeof(synInfo_t));  // 显存空间
  hipMemcpy(sInfo->nInfoDevice, sInfo->nInfoHost,
    sInfo->preN*sInfo->Ndelay*sInfo->MaxN*NTh*sizeof(synInfo_t),
    hipMemcpyHostToDevice);  // 拷贝数据
	//hipMalloc 突触数据 
	//显存变量：sInfo->sInfoDevice 对应主存变量：sInfo->sInfoHost (类型synInfo_t*)
	//空间大小(long)sInfo->preN*sInfo->Ndelay*sInfo->MaxN*NTh*sizeof(synInfo_t)
	                
	//hipMemcpy 以上主存到显存数据拷贝

  hipHostMalloc(&sInfo->firingTableHost, sInfo->gSize*sizeof(spikeTime_t));  // 内存空间
  hipMalloc(&sInfo->firingTableDevice, sInfo->gSize*sizeof(spikeTime_t));  // 显存空间
  hipMemcpy(sInfo->firingTableDevice, sInfo->firingTableHost,
    sInfo->gSize*sizeof(spikeTime_t), hipMemcpyHostToDevice);  // 拷贝数据
  // 没有清零
	//以下两个变量(局部、全局脉冲事件信息)测试HtoD通信，必须建立
	//hipMalloc 本地脉冲事件表 
	//显存变量：sInfo->firingTableDevice 对应主存变量sInfo->firingTableHost (类型spikeTime_t*)
	//空间大小sInfo->gSize*sizeof(spikeTime_t)
	//hipMemset 空间清零(可有可无)
	
  hipHostMalloc(&sInfo->firingTableAll,
    sInfo->NN*sInfo->Ndelay*sizeof(spikeTime_t));  // 内存空间
  hipMalloc(&sInfo->firingTableAllDevice,
    sInfo->NN*sInfo->Ndelay*sizeof(spikeTime_t));  // 显存空间
  hipMemcpy(sInfo->firingTableAllDevice, sInfo->firingTableAll,
    sInfo->NN*sInfo->Ndelay*sizeof(spikeTime_t), hipMemcpyHostToDevice);  // 拷贝数据
  // 没有清零
	//hipMalloc 全局脉冲事件表 
	//显存变量: sInfo->firingTableAllDevice 对应主存变量sInfo->firingTableAll (类型spikeTime_t*)
	//空间大小sInfo->NN*sInfo->Ndelay*sizeof(spikeTime_t)
	//hipMemset 空间清零(可有可无)



  checkError();  // 检测错误
	return;	
}

void freeSW(){
  // 没有释放
	return;
}
// kernel_empty 空核函数 只是启动
__global__ void kernel_empty() {
  // 没有内容
}
void StateUpdate(snnInfo_t *sInfo){
	//kernel 函数启动一次
  kernel_empty<<<128, 128>>>();
  hipDeviceSynchronize();  // 等待核函数运行结束
	//DtoH脉冲事件显存到主存传递
  hipMemcpy(sInfo->firingTableAll, sInfo->firingTableAllDevice,
    ((sInfo->gSize-1)/100+1)*sizeof(spikeTime_t), hipMemcpyDeviceToHost);  // 拷贝数据
	//显存变量: sInfo->firingTableAllDevice 对应主存变量sInfo->firingTableAll (类型spikeTime_t*)
	//传递大小: ((sInfo->gSize-1)/100+1)*sizeof(spikeTime_t)

	return;
}
void SpikeDeliver(snnInfo_t *sInfo){

  hipMemcpy(sInfo->firingTableAllDevice, sInfo->firingTableAll,
    ((sInfo->NN-1)/100+1)*sizeof(spikeTime_t), hipMemcpyHostToDevice);  // 拷贝数据

	//HtoD脉冲事件主存到显存传递
	//显存变量: sInfo->firingTableAllDevice 对应主存变量sInfo->firingTableAll (类型spikeTime_t*)
	//传递大小: ((sInfo->NN-1)/100+1)*sizeof(spikeTime_t)
	
	//kernel 函数启动一次
  kernel_empty<<<128, 128>>>();
  hipDeviceSynchronize();  // 等待核函数运行结束
	return;
}
//inline float dvdtIzh(float v, float u, float tmpI, float h);
//inline float dudtIzh(float v, float u, float a, float b, float h);
//inline float dvdtIzh(float v, float u, float tmpI, float h) {return (((0.04*v+5.0)*v+140.0-u+tmpI)*h);}
//inline float dudtIzh(float v, float u, float a, float b, float h) {return (a*(b*v-u)*h);}
